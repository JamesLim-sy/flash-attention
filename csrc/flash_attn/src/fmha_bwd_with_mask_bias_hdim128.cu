// Copyright (c) 2022, Tri Dao.

// Splitting the different head dimensions to different files to speed up compilation.

#include "fmha_bwd_launch_template.h"

void run_fmha_bwd_with_mask_bias_hdim128(FMHA_dgrad_params &params, hipStream_t stream) {
    FP16_SWITCH(params.is_bf16, ([&] {
        using Kernel_traits = FMHA_kernel_traits<128, 128, 16, 1, 8, 0x100u, elem_type>;
        run_fmha_dgrad_fp16_sm80_loop_<Kernel_traits>(params, stream);
    }));
}
