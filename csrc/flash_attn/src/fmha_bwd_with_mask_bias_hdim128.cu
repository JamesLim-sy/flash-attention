// Copyright (c) 2022, Tri Dao.

// Splitting the different head dimensions to different files to speed up compilation.

#include "fmha_bwd_launch_template.h"

void run_fmha_bwd_with_mask_bias_hdim128(const FMHA_dgrad_params &launch_params, 
                                         hipStream_t stream) {
    FP16_SWITCH(launch_params.params.is_bf16, ([&] {
        using Kernel_traits = FMHA_kernel_traits<128, 128, 16, 1, 8, 0x100u, elem_type>;
        run_fmha_dgrad_fp16_sm80_loop_<Kernel_traits>(params, stream);
    }));
}
