// Copyright (c) 2022, Tri Dao.

// Splitting the different head dimensions to different files to speed up compilation.

#include "fmha_bwd_launch_template.h"

void run_fmha_bwd_with_mask_bias_hdim32(const FMHA_dgrad_params &launch_params, 
                                        hipStream_t stream) {
    FP16_SWITCH(launch_params.params.is_bf16, ([&] {
        if( params.seqlen_k == 128 ) {
            using Kernel_traits = FMHA_kernel_traits<128, 32, 16, 1, 8, 0x08u, elem_type>;
            run_fmha_dgrad_fp16_sm80_loop_<Kernel_traits>(params, stream);
        } else if( params.seqlen_k >= 256 ) {
            using Kernel_traits = FMHA_kernel_traits<256, 32, 16, 1, 8, 0x08u, elem_type>;
            run_fmha_dgrad_fp16_sm80_loop_<Kernel_traits>(params, stream);
        }
    }));
}
